#include "hip/hip_runtime.h"
//#include "class.hpp"
//#include "force.hpp"
#include<particle_simulator.hpp>
#include "cuda_pointer.h"
#include "force_gpu_cuda.hpp"

enum{
	N_THREAD_GPU = 32,
	N_WALK_LIMIT = 1000,
	NI_LIMIT     = N_WALK_LIMIT*1000,
	NJ_LIMIT     = N_WALK_LIMIT*10000,
};

struct EpiGPU{
	float3 pos;
    float  r_search;
	int    id_walk;
};

struct EpiDev{
	float3 pos;
    float  r_search;
};

struct EpjGPU{
	float3 pos;
    float  m;
    float  r_search;
};

struct SpjGPU{
    float3 pos;
    float  m;
};

struct ForceGPU{
	float4 accp;
    int    nnb;
};

//! device pair force of Epi and Epi with linear cutoff
inline __device__ ForceGPU dev_gravity_ep_ep(
		float  eps2,
        float  rcut2,
        float  G,
		EpiDev epii,
		EpjGPU epjj,
        ForceGPU forcei)
{
	float dx = epjj.pos.x - epii.pos.x;
	float dy = epjj.pos.y - epii.pos.y;
	float dz = epjj.pos.z - epii.pos.z;

	float r2   = eps2 + dx*dx + dy*dy + dz*dz;
    float rsmin = max(epii.r_search, epjj.r_search);
    if (r2 < rsmin*rsmin) forcei.nnb ++;

    float r2_cut = (r2 > rcut2)? r2 : rcut2;
	float rinv = rsqrtf(r2_cut);
	float pij  = epjj.m * rinv;
	float mri3 = G*rinv*rinv * pij;

	forcei.accp.x += mri3 * dx;
	forcei.accp.y += mri3 * dy;
	forcei.accp.z += mri3 * dz;
	forcei.accp.w -= G*pij;

    return forcei;
}

__device__ ForceGPU force_kernel_ep_ep_1walk(
		EpjGPU       *jpsh,
		const EpiDev  epii,
		const int     id_walk,
		const int3   *ij_disp,
		const EpjGPU *epj, 
		ForceGPU      forcei,
		const float   eps2,
        const float   rcut2,
        const float   G)
{
    const int tid = threadIdx.x;
    const int j_head = ij_disp[id_walk  ].y;
    const int j_tail = ij_disp[id_walk+1].y;

	for(int j=j_head; j<j_tail; j+=N_THREAD_GPU){
		// __syncthreads();
		jpsh[tid] = ((EpjGPU *)(epj + j)) [tid];
		// __syncthreads();

		if(j_tail-j < N_THREAD_GPU){
			for(int jj=0; jj<j_tail-j; jj++){
				forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[jj], forcei);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[jj], forcei);
			}
		}
	}
	
	return forcei;
}

__device__ ForceGPU force_kernel_ep_ep_2walk(
		EpjGPU        jpsh[2][N_THREAD_GPU],
		const EpiDev  epii,
		const int     id_walk,
		const int     iwalk0,
		const int     iwalk1,
		const int3   *ij_disp,
		const EpjGPU *epj, 
		ForceGPU      forcei,
		const float   eps2,
        const float   rcut2,
        const float   G)
{
	const int jbeg0 = ij_disp[iwalk0].y;
	const int jbeg1 = ij_disp[iwalk1].y;
	const int jend0 = ij_disp[iwalk0 + 1].y;
	const int jend1 = ij_disp[iwalk1 + 1].y;
	const int nj0   = jend0 - jbeg0;
	const int nj1   = jend1 - jbeg1;

	const int nj_longer  = nj0 > nj1 ? nj0 : nj1;
	const int nj_shorter = nj0 > nj1 ? nj1 : nj0;
	const int walk_longer= nj0 > nj1 ? 0 : 1;
	const int jbeg_longer = nj0 > nj1 ? jbeg0 : jbeg1;

	const int mywalk = id_walk==iwalk0 ? 0 : 1;

    const int tid = threadIdx.x;
	for(int j=0; j<nj_shorter; j+=N_THREAD_GPU){
		jpsh[0][tid] = ((EpjGPU *)(epj + jbeg0 + j)) [tid];
		jpsh[1][tid] = ((EpjGPU *)(epj + jbeg1 + j)) [tid];
		if(nj_shorter-j < N_THREAD_GPU){
			for(int jj=0; jj<nj_shorter-j; jj++){
				forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[mywalk][jj], forcei);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[mywalk][jj], forcei);
			}
		}
	}
	for(int j=nj_shorter; j<nj_longer; j+=N_THREAD_GPU){
		jpsh[0][tid] = ((EpjGPU *)(epj + jbeg_longer +  j)) [tid];
		int jrem = nj_longer - j;
		if(jrem < N_THREAD_GPU){
			for(int jj=0; jj<jrem; jj++){
				if(mywalk == walk_longer)
                    forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[0][jj], forcei);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				if(mywalk == walk_longer)
                    forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, jpsh[0][jj], forcei);
			}
		}
	}

	return forcei;
}

__device__ ForceGPU force_kernel_ep_ep_multiwalk(
		const EpiDev  epii,
		const int     id_walk,
		const int3   *ij_disp,
		const EpjGPU *epj, 
		ForceGPU      forcei,
		const float   eps2,
        const float   rcut2,
        const float   G)
{
    const int j_head = ij_disp[id_walk  ].y;
    const int j_tail = ij_disp[id_walk+1].y;

    for(int j=j_head; j<j_tail; j++){
		EpjGPU epjj = epj[j];
		forcei = dev_gravity_ep_ep(eps2, rcut2, G, epii, epjj, forcei);
	}
	return forcei;
}

__global__ void force_kernel_ep_ep(
		const int3   * ij_disp,
		const EpiGPU * epi,
		const EpjGPU * epj, 
		ForceGPU     * force,
		const float    eps2,
        const float    rcut2,
        const float    G)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	EpiDev epii;
    epii.pos       = epi[tid].pos;
    epii.r_search  = epi[tid].r_search;
	int    id_walk = epi[tid].id_walk;
	ForceGPU forcei;
    forcei.accp = make_float4(0.f, 0.f, 0.f, 0.f);
    forcei.nnb  = 0;

	int t_head = blockDim.x * blockIdx.x;
	int t_tail = t_head + N_THREAD_GPU - 1;
	int nwalk_in_block = 1 + (epi[t_tail].id_walk - epi[t_head].id_walk);

	__shared__ EpjGPU jpsh[2][N_THREAD_GPU];

	if(1 == nwalk_in_block){
		forcei = force_kernel_ep_ep_1walk(jpsh[0], epii, id_walk, ij_disp, epj, forcei, eps2, rcut2, G);
	} else if(2 == nwalk_in_block){
		// accp = force_kernel_ep_ep_multiwalk(epii, id_walk, ij_disp, epj, accp, eps2);
		int iwalk0 = epi[t_head].id_walk;
		int iwalk1 = epi[t_tail].id_walk;
		forcei = force_kernel_ep_ep_2walk(jpsh, epii, id_walk, iwalk0, iwalk1, ij_disp, epj, forcei, eps2, rcut2, G);
	} else{
		forcei = force_kernel_ep_ep_multiwalk(epii, id_walk, ij_disp, epj, forcei, eps2, rcut2, G);
	}
	force[tid] = forcei;
}

//! device pair force of Epi and Spi 
inline __device__ float4 dev_gravity_ep_sp(
		float  eps2,
        float  G,
		float3 posi,
		SpjGPU spjj,
        float4 accpi)
{
	float dx = spjj.pos.x - posi.x;
	float dy = spjj.pos.y - posi.y;
	float dz = spjj.pos.z - posi.z;

	float r2   = eps2 + dx*dx + dy*dy + dz*dz;
	float rinv = rsqrtf(r2);
	float pij  = spjj.m * rinv;
	float mri3 = G*rinv*rinv * pij;

	accpi.x += mri3 * dx;
	accpi.y += mri3 * dy;
	accpi.z += mri3 * dz;
	accpi.w -= G*pij;

    return accpi;
}

__device__ float4 force_kernel_ep_sp_1walk(
		SpjGPU   *jpsh,
		const float3  posi,
		const int     id_walk,
		const int3   *ij_disp,
		const SpjGPU *spj, 
		float4        accpi,
		const float   eps2,
        const float   G)
{
    const int tid = threadIdx.x;
    const int j_head = ij_disp[id_walk  ].z;
    const int j_tail = ij_disp[id_walk+1].z;

	for(int j=j_head; j<j_tail; j+=N_THREAD_GPU){
		// __syncthreads();
		jpsh[tid] = ((SpjGPU *)(spj + j)) [tid];
		// __syncthreads();

		if(j_tail-j < N_THREAD_GPU){
			for(int jj=0; jj<j_tail-j; jj++){
				accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[jj], accpi);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[jj], accpi);
			}
		}
	}
	
	return accpi;
}

__device__ float4 force_kernel_ep_sp_2walk(
		SpjGPU        jpsh[2][N_THREAD_GPU],
		const float3  posi,
		const int     id_walk,
		const int     iwalk0,
		const int     iwalk1,
		const int3   *ij_disp,
		const SpjGPU *spj, 
		float4        accpi,
		const float   eps2,
        const float   G)
{
	const int jbeg0 = ij_disp[iwalk0].z;
	const int jbeg1 = ij_disp[iwalk1].z;
	const int jend0 = ij_disp[iwalk0 + 1].z;
	const int jend1 = ij_disp[iwalk1 + 1].z;
	const int nj0   = jend0 - jbeg0;
	const int nj1   = jend1 - jbeg1;

	const int nj_longer  = nj0 > nj1 ? nj0 : nj1;
	const int nj_shorter = nj0 > nj1 ? nj1 : nj0;
	const int walk_longer= nj0 > nj1 ? 0 : 1;
	const int jbeg_longer = nj0 > nj1 ? jbeg0 : jbeg1;

	const int mywalk = id_walk==iwalk0 ? 0 : 1;

    const int tid = threadIdx.x;
	for(int j=0; j<nj_shorter; j+=N_THREAD_GPU){
		jpsh[0][tid] = ((SpjGPU *)(spj + jbeg0 + j)) [tid];
		jpsh[1][tid] = ((SpjGPU *)(spj + jbeg1 + j)) [tid];
		if(nj_shorter-j < N_THREAD_GPU){
			for(int jj=0; jj<nj_shorter-j; jj++){
				accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[mywalk][jj], accpi);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[mywalk][jj], accpi);
			}
		}
	}
	for(int j=nj_shorter; j<nj_longer; j+=N_THREAD_GPU){
		jpsh[0][tid] = ((SpjGPU *)(spj + jbeg_longer +  j)) [tid];
		int jrem = nj_longer - j;
		if(jrem < N_THREAD_GPU){
			for(int jj=0; jj<jrem; jj++){
				if(mywalk == walk_longer)
                    accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[0][jj], accpi);
			}
		}else{
#pragma unroll
			for(int jj=0; jj<N_THREAD_GPU; jj++){
				if(mywalk == walk_longer)
                    accpi = dev_gravity_ep_sp(eps2, G, posi, jpsh[0][jj], accpi);
			}
		}
	}

	return accpi;
}

__device__ float4 force_kernel_ep_sp_multiwalk(
		const float3  posi,
		const int     id_walk,
		const int3   *ij_disp,
		const SpjGPU *spj, 
		float4        accpi,
		const float   eps2,
        const float   G)
{
    const int j_head = ij_disp[id_walk  ].z;
    const int j_tail = ij_disp[id_walk+1].z;

    for(int j=j_head; j<j_tail; j++){
		SpjGPU spjj = spj[j];
		accpi = dev_gravity_ep_sp(eps2, G, posi, spjj, accpi);
	}
	return accpi;
}

__global__ void force_kernel_ep_sp(
		const int3   * ij_disp,
		const EpiGPU * epi,
		const SpjGPU * spj, 
		ForceGPU     * force,
		const float    eps2,
        const float    G)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	float3 posi    = epi[tid].pos;
	int    id_walk = epi[tid].id_walk;
	float4 accpi   = force[tid].accp;

	int t_head = blockDim.x * blockIdx.x;
	int t_tail = t_head + N_THREAD_GPU - 1;
	int nwalk_in_block = 1 + (epi[t_tail].id_walk - epi[t_head].id_walk);

	__shared__ SpjGPU jpsh[2][N_THREAD_GPU];

	if(1 == nwalk_in_block){
		accpi = force_kernel_ep_sp_1walk(jpsh[0], posi, id_walk, ij_disp, spj, accpi, eps2, G);
	} else if(2 == nwalk_in_block){
		// accpi = force_kernel_ep_sp_multiwalk(posi, id_walk, ij_disp, spj, accpi, eps2);
		int iwalk0 = epi[t_head].id_walk;
		int iwalk1 = epi[t_tail].id_walk;
		accpi = force_kernel_ep_sp_2walk(jpsh, posi, id_walk, iwalk0, iwalk1, ij_disp, spj, accpi, eps2, G);
	} else{
		accpi = force_kernel_ep_sp_multiwalk(posi, id_walk, ij_disp, spj, accpi, eps2, G);
	}
	force[tid].accp = accpi;
}

static cudaPointer<EpiGPU>    dev_epi;
static cudaPointer<EpjGPU>    dev_epj;
static cudaPointer<SpjGPU>    dev_spj;
static cudaPointer<ForceGPU>  dev_force;
static cudaPointer<int3>      ij_disp;
static bool init_call = true;

PS::S32 DispatchKernelWithSP(const PS::S32          tag,
                             const PS::S32          n_walk,
                             const EPISoft         *epi[],
                             const PS::S32          n_epi[],
                             const EPJSoft          *epj[],
                             const PS::S32          n_epj[],
                             const PS::SPJMonopoleInAndOut *spj[],
                             const PS::S32          n_spj[]){
    assert(n_walk <= N_WALK_LIMIT);
    if(init_call){
		dev_epi  .allocate(NI_LIMIT);
		dev_epj  .allocate(NJ_LIMIT);
        dev_spj  .allocate(NJ_LIMIT);
		dev_force.allocate(NI_LIMIT);
		ij_disp  .allocate(N_WALK_LIMIT+2);
		init_call = false;
    }
    const float eps2 = EPISoft::eps * EPISoft::eps;
    const PS::F64 rcut2 = EPISoft::r_out*EPISoft::r_out;
    const PS::F64 G = ForceSoft::grav_const;
    ij_disp[0].x = 0;
    ij_disp[0].y = 0;
    ij_disp[0].z = 0;
    for(int k=0; k<n_walk; k++){
        ij_disp[k+1].x = ij_disp[k].x + n_epi[k];
        ij_disp[k+1].y = ij_disp[k].y + n_epj[k];
        ij_disp[k+1].z = ij_disp[k].z + n_spj[k];
    }
    ij_disp[n_walk+1] = ij_disp[n_walk];

    assert(ij_disp[n_walk].x < NI_LIMIT);
    assert(ij_disp[n_walk].y < NJ_LIMIT);
    assert(ij_disp[n_walk].z < NJ_LIMIT);
    ij_disp.htod(n_walk + 2);

    int ni_tot_reg = ij_disp[n_walk].x;
    if(ni_tot_reg % N_THREAD_GPU){
        ni_tot_reg /= N_THREAD_GPU;
        ni_tot_reg++;
        ni_tot_reg *= N_THREAD_GPU;
    }

    int ni_tot = 0;
    int nej_tot = 0;
    int nsj_tot = 0;
    for(int iw=0; iw<n_walk; iw++){
        for(int i=0; i<n_epi[iw]; i++){
            dev_epi[ni_tot].pos.x = epi[iw][i].pos.x;
            dev_epi[ni_tot].pos.y = epi[iw][i].pos.y;
            dev_epi[ni_tot].pos.z = epi[iw][i].pos.z;
            dev_epi[ni_tot].r_search = epi[iw][i].r_search;
            dev_epi[ni_tot].id_walk = iw;
            ni_tot++;
        }
        for(int j=0; j<n_epj[iw]; j++){
            dev_epj[nej_tot].pos.x  = epj[iw][j].pos.x;
            dev_epj[nej_tot].pos.y  = epj[iw][j].pos.y;
            dev_epj[nej_tot].pos.z  = epj[iw][j].pos.z;
            dev_epj[nej_tot].m      = epj[iw][j].mass;
            dev_epj[nej_tot].r_search = epj[iw][j].r_search;
            nej_tot++;
        }
        for(int j=0; j<n_spj[iw]; j++){
            dev_spj[nsj_tot].pos.x  = spj[iw][j].pos.x;
            dev_spj[nsj_tot].pos.y  = spj[iw][j].pos.y;
            dev_spj[nsj_tot].pos.z  = spj[iw][j].pos.z;
            dev_spj[nsj_tot].m      = spj[iw][j].getCharge();
            nsj_tot++;
        }
    }
    for(int i=ni_tot; i<ni_tot_reg; i++){
        dev_epi[i].id_walk = n_walk;
    }

    dev_epi.htod(ni_tot_reg);
    dev_epj.htod(nej_tot);
    dev_spj.htod(nsj_tot);

    int nblocks  = ni_tot_reg / N_THREAD_GPU;
    int nthreads = N_THREAD_GPU;
    force_kernel_ep_ep <<<nblocks, nthreads>>> (ij_disp, dev_epi, dev_epj, dev_force, eps2, rcut2, G);
    force_kernel_ep_sp <<<nblocks, nthreads>>> (ij_disp, dev_epi, dev_spj, dev_force, eps2, G);

    return 0;
}

PS::S32 RetrieveKernel(const PS::S32 tag,
                       const PS::S32 n_walk,
                       const PS::S32 ni[],
                       ForceSoft    *force[])
{
    int ni_tot = 0;
    for(int k=0; k<n_walk; k++){
        ni_tot += ni[k];
    }
    dev_force.dtoh(ni_tot);

    int n_cnt = 0;
    for(int iw=0; iw<n_walk; iw++){
        for(int i=0; i<ni[iw]; i++){
            force[iw][i].acc.x = dev_force[n_cnt].accp.x;
            force[iw][i].acc.y = dev_force[n_cnt].accp.y;
            force[iw][i].acc.z = dev_force[n_cnt].accp.z;
            force[iw][i].pot   = dev_force[n_cnt].accp.w;
            force[iw][i].n_ngb = dev_force[n_cnt].nnb;
            n_cnt++;
        }
    }
    return 0;
}
